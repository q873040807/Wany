#include<hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<stdlib.h>
__global__ void Comp(int**ctable1,int **ctable2,int **cresult,int j_N)
{
	int i=threadIdx.x;
	int j=threadIdx.y;
	for(int z=0;z<j_N;z++)
		cresult[i][j]+=ctable1[i][z]*ctable2[z][j];
}
int** MallocDeviceDoubleArray(int** head,int Array_i,int Array_j);
int** MallocHostDoubleArray(int Array_i,int Array_j);
int main()
{
	int i1_max,j1_max,i2_max,j2_max;
	printf("����һ��������˵ĳ���\n");
	printf("���������A��С(���д�С)\n");
	scanf("%d %d",&j1_max,&i1_max);	//���þ���Ĵ�С
	int **table1=MallocHostDoubleArray(i1_max,j1_max);
	printf("���������A������\n");
	for(int i=0;i<i1_max;i++)
		for(int j=0;j<j1_max;j++)
			scanf("%d",&table1[i][j]);

	printf("���������B��С(���д�С)\n");
	scanf("%d %d",&j2_max,&i2_max);	//���þ���Ĵ�С
	int **table2=MallocHostDoubleArray(i2_max,j2_max);	//Ϊ�����������host�ռ�
	printf("���������B������\n");
	for(int i=0;i<i2_max;i++)
		for(int j=0;j<j2_max;j++)
			scanf("%d",&table2[i][j]);

	/*
	for(int i=0;i<i1_max;i++)
	{
	for(int j=0;j<j1_max;j++)
	printf("%d ",table1[i][j]);
	printf("\n");
	}

	for(int i=0;i<i2_max;i++)
	{
	for(int j=0;j<j2_max;j++)
	printf("%d ",table2[i][j]);
	printf("\n");
	}*/


	int **result=MallocHostDoubleArray(i1_max,j2_max);		//���䴢������host�ռ�
	int *ctable1_head,*ctable2_head,*cresult_head;
	int **ctable1=MallocDeviceDoubleArray(&ctable1_head,i1_max,j1_max),**ctable2=MallocDeviceDoubleArray(&ctable2_head,i2_max,j2_max),**cresult=MallocDeviceDoubleArray(&cresult_head,i1_max,j2_max);	//Ϊ������������Դ�ռ�
	hipMemset(cresult_head,0,sizeof(cresult_head));
	hipMemcpy(ctable1_head,*table1,sizeof(int)*i1_max*j1_max,hipMemcpyHostToDevice);
	hipMemcpy(ctable2_head,*table2,sizeof(int)*i2_max*j2_max,hipMemcpyHostToDevice);		//Ϊtable1��2��ֵ���Դ�
	dim3 threadmax;
	threadmax.x=i1_max;
	threadmax.y=j2_max;
	Comp<<<1,threadmax>>>(ctable1,ctable2,cresult,j1_max);
	hipMemcpy(*result,cresult_head,sizeof(int)*i1_max*j2_max,hipMemcpyDeviceToHost);
	for(int i=0;i<i1_max;i++)
	{
		for(int j=0;j<j2_max;j++)
			printf("%d ",result[i][j]);
		printf("\n");	
	}
	return 0;
}
/*
*@�ö�ά�������ʽ����һά����
*@����ά������ʣ�һά����洢��
*@Device�����ɶ�̬��ά���飬head����һά����ĵ�ַ���Ա�memcpyʹ�ã�
*@����ֵΪ��ά����ָ��
*/
int** MallocDeviceDoubleArray(int** head,int Array_i,int Array_j)
{
	int** cresult,*temp,**temp3;
	hipMalloc((void**)&cresult,sizeof(int*)*Array_i);
	hipMalloc((void**)&temp,sizeof(int)*Array_i*Array_j);
	temp3=(int**)malloc(sizeof(int*)*Array_i);
	for(int i=0;i<Array_i;i++)
		temp3[i]=i*Array_j+temp;
	*head=temp;
	hipMemcpy(cresult,temp3,sizeof(int*)*Array_i,hipMemcpyHostToDevice);
	return cresult;
}
int **MallocHostDoubleArray(int Array_i,int Array_j)
{
	int **table,*table2;
	table2=(int*)malloc(sizeof(int)*Array_i*Array_j);
	table=(int**)malloc(sizeof(int*)*Array_i);
	for(int i=0;i<Array_i;i++)
	{
		table[i]=Array_j*i+table2;
	}
	return table;
}