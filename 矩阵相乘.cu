#include<hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<stdlib.h>
__global__ void Comp(int**ctable1,int **ctable2,int **cresult,int j_N)
{
	int i=threadIdx.x;
	int j=threadIdx.y;
	for(int z=0;z<j_N;z++)
		cresult[i][j]+=ctable1[i][z]*ctable2[z][j];
}
int** MallocDeviceDoubleArray(int** head,int Array_i,int Array_j);
int** MallocHostDoubleArray(int Array_i,int Array_j);
int main()
{
	int i1_max,j1_max,i2_max,j2_max;
	printf("这是一个矩阵相乘的程序\n");
	printf("请输入矩阵A大小(行列大小)\n");
	scanf("%d %d",&j1_max,&i1_max);	//设置矩阵的大小
	int **table1=MallocHostDoubleArray(i1_max,j1_max);
	printf("请输入矩阵A的数据\n");
	for(int i=0;i<i1_max;i++)
		for(int j=0;j<j1_max;j++)
			scanf("%d",&table1[i][j]);

	printf("请输入矩阵B大小(行列大小)\n");
	scanf("%d %d",&j2_max,&i2_max);	//设置矩阵的大小
	int **table2=MallocHostDoubleArray(i2_max,j2_max);	//为两个矩阵分配host空间
	printf("请输入矩阵B的数据\n");
	for(int i=0;i<i2_max;i++)
		for(int j=0;j<j2_max;j++)
			scanf("%d",&table2[i][j]);

	/*
	for(int i=0;i<i1_max;i++)
	{
	for(int j=0;j<j1_max;j++)
	printf("%d ",table1[i][j]);
	printf("\n");
	}

	for(int i=0;i<i2_max;i++)
	{
	for(int j=0;j<j2_max;j++)
	printf("%d ",table2[i][j]);
	printf("\n");
	}*/


	int **result=MallocHostDoubleArray(i1_max,j2_max);		//分配储存结果的host空间
	int *ctable1_head,*ctable2_head,*cresult_head;
	int **ctable1=MallocDeviceDoubleArray(&ctable1_head,i1_max,j1_max),**ctable2=MallocDeviceDoubleArray(&ctable2_head,i2_max,j2_max),**cresult=MallocDeviceDoubleArray(&cresult_head,i1_max,j2_max);	//为两个矩阵分配显存空间
	hipMemset(cresult_head,0,sizeof(cresult_head));
	hipMemcpy(ctable1_head,*table1,sizeof(int)*i1_max*j1_max,hipMemcpyHostToDevice);
	hipMemcpy(ctable2_head,*table2,sizeof(int)*i2_max*j2_max,hipMemcpyHostToDevice);		//为table1，2赋值到显存
	dim3 threadmax;
	threadmax.x=i1_max;
	threadmax.y=j2_max;
	Comp<<<1,threadmax>>>(ctable1,ctable2,cresult,j1_max);
	hipMemcpy(*result,cresult_head,sizeof(int)*i1_max*j2_max,hipMemcpyDeviceToHost);
	for(int i=0;i<i1_max;i++)
	{
		for(int j=0;j<j2_max;j++)
			printf("%d ",result[i][j]);
		printf("\n");	
	}
	return 0;
}
/*
*@用二维数组的形式访问一维数组
*@（二维数组访问，一维数组存储）
*@Device端生成动态二维数组，head返回一维数组的地址（以便memcpy使用）
*@返回值为二维数组指针
*/
int** MallocDeviceDoubleArray(int** head,int Array_i,int Array_j)
{
	int** cresult,*temp,**temp3;
	hipMalloc((void**)&cresult,sizeof(int*)*Array_i);
	hipMalloc((void**)&temp,sizeof(int)*Array_i*Array_j);
	temp3=(int**)malloc(sizeof(int*)*Array_i);
	for(int i=0;i<Array_i;i++)
		temp3[i]=i*Array_j+temp;
	*head=temp;
	hipMemcpy(cresult,temp3,sizeof(int*)*Array_i,hipMemcpyHostToDevice);
	return cresult;
}
int **MallocHostDoubleArray(int Array_i,int Array_j)
{
	int **table,*table2;
	table2=(int*)malloc(sizeof(int)*Array_i*Array_j);
	table=(int**)malloc(sizeof(int*)*Array_i);
	for(int i=0;i<Array_i;i++)
	{
		table[i]=Array_j*i+table2;
	}
	return table;
}
