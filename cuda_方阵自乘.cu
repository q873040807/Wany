#include<hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<stdlib.h>
__global__ void Comp(int**ctable,int**result,int i_N)
{
	int i=threadIdx.x;
	int j=threadIdx.y;
	for(int z=0;z<i_N;z++)
		result[i][j]+=ctable[z][j]*ctable[i][z];
}
int** MallocDeviceDoubleArray(int** head,int Array_i,int Array_j);
int** MallocHostDoubleArray(int Array_i,int Array_j);
int main()
{
	int i_N,j_N;
	printf("请输入矩阵的大小 i,j\n");
	scanf("%d %d",&j_N,&i_N);
	int** table=MallocHostDoubleArray(i_N,j_N);
	printf("请输入矩阵元素\n");
	for(int i=0;i<i_N;i++)
		for(int j=0;j<j_N;j++)
			scanf("%d",&table[i][j]);
	int** result=MallocHostDoubleArray( i_N,j_N);
	int* cresult_head,*ctable_head;
	int** cresult =MallocDeviceDoubleArray(&cresult_head,i_N,j_N),**ctable=MallocDeviceDoubleArray(&ctable_head,i_N,j_N);
	hipMemset(cresult_head,0,sizeof(cresult_head));
	hipMemcpy(ctable_head,*table,sizeof(int)*i_N*j_N,hipMemcpyHostToDevice);
	dim3 threadsize;
	threadsize.x=i_N;
	threadsize.y=j_N;
	Comp<<<1,threadsize>>>(ctable,cresult,i_N);
	hipMemcpy(*result,cresult_head,sizeof(int)*i_N*j_N,hipMemcpyDeviceToHost);
	for(int i=0;i<i_N;i++)
	{
		for(int j=0;j<j_N;j++)
			printf("%6d ",result[i][j]);
		printf("\n");
	}
	return 0;
}
/*
*@用二维数组的形式访问一维数组
*@（二维数组访问，一维数组存储）
*@Device端生成动态二维数组，head返回一维数组的地址（以便memcpy使用）
*@返回值为二维数组指针
*/
int** MallocDeviceDoubleArray(int** head,int Array_i,int Array_j)
{
	int** cresult,*temp,**temp3;
	hipMalloc((void**)&cresult,sizeof(int*)*Array_i);
	hipMalloc((void**)&temp,sizeof(int)*Array_i*Array_j);
	temp3=(int**)malloc(sizeof(int*)*Array_i);
	for(int i=0;i<Array_i;i++)
		temp3[i]=i*Array_j+temp;
	*head=temp;
	hipMemcpy(cresult,temp3,sizeof(int*)*Array_i,hipMemcpyHostToDevice);
	return cresult;
}
int **MallocHostDoubleArray(int Array_i,int Array_j)
{
	int **table,*table2;
	table2=(int*)malloc(sizeof(int)*Array_i*Array_j);
	table=(int**)malloc(sizeof(int*)*Array_i);
	for(int i=0;i<Array_i;i++)
	{
		table[i]=Array_j*i+table2;
	}
	return table;
}