#include<hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<stdlib.h>
__global__ void Comp(int**ctable,int**result,int i_N)
{
	int i=threadIdx.x;
	int j=threadIdx.y;
	for(int z=0;z<i_N;z++)
		result[i][j]+=ctable[z][j]*ctable[i][z];
}
int** MallocDeviceDoubleArray(int** head,int Array_i,int Array_j);
int** MallocHostDoubleArray(int Array_i,int Array_j);
int main()
{
	int i_N,j_N;
	printf("���������Ĵ�С i,j\n");
	scanf("%d %d",&j_N,&i_N);
	int** table=MallocHostDoubleArray(i_N,j_N);
	printf("���������Ԫ��\n");
	for(int i=0;i<i_N;i++)
		for(int j=0;j<j_N;j++)
			scanf("%d",&table[i][j]);
	int** result=MallocHostDoubleArray( i_N,j_N);
	int* cresult_head,*ctable_head;
	int** cresult =MallocDeviceDoubleArray(&cresult_head,i_N,j_N),**ctable=MallocDeviceDoubleArray(&ctable_head,i_N,j_N);
	hipMemset(cresult_head,0,sizeof(cresult_head));
	hipMemcpy(ctable_head,*table,sizeof(int)*i_N*j_N,hipMemcpyHostToDevice);
	dim3 threadsize;
	threadsize.x=i_N;
	threadsize.y=j_N;
	Comp<<<1,threadsize>>>(ctable,cresult,i_N);
	hipMemcpy(*result,cresult_head,sizeof(int)*i_N*j_N,hipMemcpyDeviceToHost);
	for(int i=0;i<i_N;i++)
	{
		for(int j=0;j<j_N;j++)
			printf("%6d ",result[i][j]);
		printf("\n");
	}
	return 0;
}
/*
*@�ö�ά�������ʽ����һά����
*@����ά������ʣ�һά����洢��
*@Device�����ɶ�̬��ά���飬head����һά����ĵ�ַ���Ա�memcpyʹ�ã�
*@����ֵΪ��ά����ָ��
*/
int** MallocDeviceDoubleArray(int** head,int Array_i,int Array_j)
{
	int** cresult,*temp,**temp3;
	hipMalloc((void**)&cresult,sizeof(int*)*Array_i);
	hipMalloc((void**)&temp,sizeof(int)*Array_i*Array_j);
	temp3=(int**)malloc(sizeof(int*)*Array_i);
	for(int i=0;i<Array_i;i++)
		temp3[i]=i*Array_j+temp;
	*head=temp;
	hipMemcpy(cresult,temp3,sizeof(int*)*Array_i,hipMemcpyHostToDevice);
	return cresult;
}
int **MallocHostDoubleArray(int Array_i,int Array_j)
{
	int **table,*table2;
	table2=(int*)malloc(sizeof(int)*Array_i*Array_j);
	table=(int**)malloc(sizeof(int*)*Array_i);
	for(int i=0;i<Array_i;i++)
	{
		table[i]=Array_j*i+table2;
	}
	return table;
}